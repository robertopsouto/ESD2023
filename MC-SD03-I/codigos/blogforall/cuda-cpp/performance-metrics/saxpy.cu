#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char* argv[])
{

  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <matrix size N> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);
  int devId = atoi(argv[2]);

  printf("Number of Elements : %d\n", N);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devId);
  printf("Device : %s\n", prop.name);
  hipSetDevice(devId);

  float *x, *y;
  // allocate the memory on the CPU
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  float *d_x, *d_y;
  // allocate the memory on the GPU
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(start);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);

  hipEventRecord(stop);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
  }

  printf("Max error: %f\n", maxError);
  printf("Execution Time (milliseconds): %f\n", milliseconds);
  printf("Effective Bandwidth (GB/s): %f\n", N*4*3/milliseconds*1e3/1e9);
  printf("Effective Performance (GFLOP/s): %f\n", N*2/milliseconds*1e3/1e9);

  // free the memory allocated on the CPU
  free(x); x=NULL;
  free(y); y=NULL;

  // free the memory allocated on the GPU
  hipFree( d_x );
  hipFree( d_y );
}
