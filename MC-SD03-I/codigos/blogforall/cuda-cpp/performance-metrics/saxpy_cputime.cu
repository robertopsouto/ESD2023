#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(int argc, char* argv[])
{

  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <matrix size N> <block size> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);
  int devId = atoi(argv[2]);

  printf("Number of Elements : %d\n", N);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, devId);
  printf("Device : %s\n", prop.name);
  hipSetDevice(devId);

  float *x, *y;
  // allocate the memory on the CPU
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));

  float *d_x, *d_y;
  // allocate the memory on the GPU
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  struct timeval begin, end;
  gettimeofday(&begin, NULL);
  
  // Perform SAXPY on 1M elements
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);

  hipDeviceSynchronize();
  gettimeofday(&end, NULL);
  double cpuTime = 1000000*(double)(end.tv_sec - begin.tv_sec);
  cpuTime +=  (double)(end.tv_usec - begin.tv_usec);
  
  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = max(maxError, abs(y[i]-4.0f));
  }

  printf("Max error: %f\n", maxError);
  printf("Execution Time (miliseconds): %f\n", cpuTime/1000.0);
  printf("Effective Bandwidth (GB/s): %f\n", N*4*3/cpuTime*1e6/1e9);
  printf("Effective Performance (GFLOP/s): %f\n", N*2/cpuTime*1e6/1e9);

  // free the memory allocated on the CPU
  free(x); x=NULL;
  free(y); y=NULL;

  // free the memory allocated on the GPU
  hipFree( d_x );
  hipFree( d_y );
}
