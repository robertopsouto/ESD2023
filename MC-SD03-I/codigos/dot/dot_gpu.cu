/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


#define imin(a,b) (a<b?a:b)

const int BlockSize = 256;

__global__ void dot( float *a, float *b, float *c, int N ) {
    __shared__ float cache[BlockSize];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float  temp = 0; 
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, BlockSize must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main(int argc, char* argv[])
{

    //printf("Argumentos: %d \n",argc);

    if (argc != 3) {
      fprintf(stderr, "Syntax: %s <vector size N> <device id>\n", argv[0]);
      return EXIT_FAILURE;
    }

    int N = atoi(argv[1]);
    int GridSize = imin( 32, (N+BlockSize-1) / BlockSize );
    int devId = atoi(argv[2]);

    checkCuda( hipSetDevice(devId) );

    float   *a, *b, c, *partial_c;
    float   *dev_a, *dev_b, *dev_partial_c;

    // allocate memory on the cpu side
    a = (float*)malloc( N*sizeof(float) );
    b = (float*)malloc( N*sizeof(float) );
    partial_c = (float*)malloc( GridSize*sizeof(float) );

    // allocate the memory on the GPU
    checkCuda( hipMalloc( (void**)&dev_a, N*sizeof(float) ) );
    checkCuda( hipMalloc( (void**)&dev_b, N*sizeof(float) ) );
    checkCuda( hipMalloc( (void**)&dev_partial_c, GridSize*sizeof(float) ) );

    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // copy the arrays 'a' and 'b' to the GPU
    checkCuda( hipMemcpy( dev_a, a, N*sizeof(float), hipMemcpyHostToDevice ) );
    checkCuda( hipMemcpy( dev_b, b, N*sizeof(float), hipMemcpyHostToDevice ) ); 

    dot<<<GridSize,BlockSize>>>( dev_a, dev_b, dev_partial_c, N );

    // copy the array 'c' back from the GPU to the CPU
    checkCuda( hipMemcpy( partial_c, dev_partial_c, GridSize*sizeof(float), hipMemcpyDeviceToHost ) );

    // finish up on the CPU side
    c = 0;
    for (int i=0; i<GridSize; i++) {
        c += partial_c[i];
    }

    #define sum_squares(x)  (x*(x+1)*(2*x+1)/6)
    printf( "Does GPU value %.6g = %.6g?\n", c, 2 * sum_squares( (float)(N - 1) ) );

    // free memory on the gpu side
    checkCuda( hipFree( dev_a ) );
    checkCuda( hipFree( dev_b ) );
    checkCuda( hipFree( dev_partial_c ) );

    // free memory on the cpu side
    free( a );
    free( b );
    free( partial_c );
}
