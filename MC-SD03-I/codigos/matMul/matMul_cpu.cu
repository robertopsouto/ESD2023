
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>


// Computes the matrix product using line matrices:
void matMul(float* P, float* M, float* N, unsigned int Width) {
  for (unsigned int i = 0; i < Width; ++i) {
    for (unsigned int j = 0; j < Width; ++j) {
      P[i * Width + j] = 0.0;
      for (unsigned int k = 0; k < Width; ++k) {
        P[i * Width + j] += M[i * Width + k] * N[k * Width + j];
      }
    }
  }
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int i = 0; i < size; ++i) {
     data[i] = (float)drand48();
  }
}


int main(int argc, char* argv[])
{
  if (argc != 2) {
    fprintf(stderr, "Syntax: %s <matrix Width>\n", argv[0]);
    return EXIT_FAILURE;
  }
  int Width = atoi(argv[1]);

  // allocate host memory for matrices M and N
  printf("Allocate memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));

  // set seed for drand48()
  srand48(42);

  // initialize matrices
  printf("Initialize matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  printf("Multiply matrices...\n");
  struct timeval begin, end;
  gettimeofday(&begin, NULL);
  matMul( P, M, N, Width );
  gettimeofday(&end, NULL);

  double cpuTime = 1000000*(double)(end.tv_sec - begin.tv_sec);
  cpuTime +=  (double)(end.tv_usec - begin.tv_usec);

  // print times
  printf("\nExecution Time (microseconds): %9.2f\n", cpuTime);

  // print result
  FILE *ptr_file;
  ptr_file =fopen("matMul_cpu.out", "w");
  if (!ptr_file) return 1;

  for (int i=0; i < Width; i++){
      for (int j=0; j < Width; j++) fprintf(ptr_file,"%6.2f ", P[i * Width + j]);
      fprintf(ptr_file,"\n");
  }
  fclose(ptr_file);

  // clean up memory
  free(M);
  free(N);
  free(P);

  return 0;
}

