
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define Tile_Width 16

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width) {
  __shared__ float Mds[Tile_Width][Tile_Width];
  __shared__ float Nds[Tile_Width][Tile_Width];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the M element to work on
  int Col = blockIdx.x * Tile_Width + tx;
  int Row = blockIdx.y * Tile_Width + ty;

  float Pvalue = 0;
  // Loop over the N and P tiles required to compute the M element
  for (int m = 0; m < Width/Tile_Width; ++m) {
    // Coolaborative loading of N and P tiles into shared memory
    Mds[ty][tx] = Md[Row*Width + (m*Tile_Width + tx)];
    Nds[ty][tx] = Nd[Col + (m*Tile_Width + ty)*Width];
    __syncthreads();

    for (int k = 0; k < Tile_Width; ++k)
      Pvalue += Mds[ty][k] * Nds[k][tx];
    __syncthreads();
  }
  Pd[Row * Width + Col] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{

  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <matrix size Width> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int Width = atoi(argv[1]);
  int devId = atoi(argv[2]);

  checkCuda( hipSetDevice(devId) );

  // allocate host memory for matrices M and N
  printf("Allocate host memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));

  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  checkCuda( hipMalloc((void**) &Md, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Nd, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Pd, Width * Width * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");

  int GridSize = (Width + Tile_Width-1) / Tile_Width;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);

  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width);

  // copy result from device to host
  checkCuda( hipMemcpy( P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);

  // print result
  FILE *ptr_file;
  ptr_file =fopen("matMul_gpu_sharedmem.out", "w");
  if (!ptr_file) return 1;

  for (int ty=0; ty < Width; ty++){
      for (int tx=0; tx < Width; tx++) fprintf(ptr_file,"%6.2f ", P[ty * Width + tx]);
      fprintf(ptr_file,"\n");
  }
  fclose(ptr_file);


  // clean up memory
  free(M);
  free(N);
  free(P);
  checkCuda( hipFree(Md) );
  checkCuda( hipFree(Nd) );
  checkCuda( hipFree(Pd) );

  return 0;
}

