
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define Tile_Width 16

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matMul(float* Pd, float* Md, float* Nd, int Width) {
  float Pvalue = 0.0;

  int j = blockIdx.x * Tile_Width + threadIdx.x;
  int i = blockIdx.y * Tile_Width + threadIdx.y;

  for (int k = 0; k < Width; ++k) {
    Pvalue += Md[j * Width + k] * Nd[k * Width + i];
  }

  Pd[j * Width + i] = Pvalue;
}


// Allocates a matrix with random float entries.
void randomInit(float* data, int size) {
  for (int k = 0; k < size; ++k) {
     data[k] = (float)drand48();
  }
}

int main(int argc, char* argv[])
{

  if (argc != 3) {
    fprintf(stderr, "Syntax: %s <matrix size Width> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int Width = atoi(argv[1]);
  int devId = atoi(argv[2]);

  checkCuda( hipSetDevice(devId) );

  // allocate host memory for matrices M and N
  printf("Allocate host memory for matrices M and N...\n");
  float* M = (float*) malloc(Width * Width * sizeof(float));
  float* N = (float*) malloc(Width * Width * sizeof(float));
  float* P = (float*) malloc(Width * Width * sizeof(float));

  // set seed for drand48()
  srand48(42);

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(M, Width*Width);
  randomInit(N, Width*Width);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* Md = NULL; 
  float* Nd = NULL;
  float* Pd = NULL;
  checkCuda( hipMalloc((void**) &Md, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Nd, Width * Width * sizeof(float)) );
  checkCuda( hipMalloc((void**) &Pd, Width * Width * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(Md, M, Width*Width*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(Nd, N, Width*Width*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");

  int GridSize = (Width + Tile_Width-1) / Tile_Width;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(Tile_Width, Tile_Width);

  matMul<<< gridDim, blockDim >>>(Pd, Md, Nd, Width);

  // copy result from device to host
  checkCuda( hipMemcpy( P, Pd, Width * Width * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);

  // print result
  FILE *ptr_file;
  ptr_file =fopen("matMul_gpu_globalmem_uncoalesced.out", "w");
  if (!ptr_file) return 1;

  for (int i=0; i < Width; i++){
      for (int j=0; j < Width; j++) fprintf(ptr_file,"%6.2f ", P[i * Width + j]);
      fprintf(ptr_file,"\n");
  }
  fclose(ptr_file);


  // clean up memory
  free(M);
  free(N);
  free(P);
  checkCuda( hipFree(Md) );
  checkCuda( hipFree(Nd) );
  checkCuda( hipFree(Pd) );

  return 0;
}

