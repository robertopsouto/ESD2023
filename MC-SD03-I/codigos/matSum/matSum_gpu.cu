
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


__global__ void matSum(float* S, float* A, float* B, int N) {
  int i = blockIdx.y*blockDim.y + threadIdx.y;
  int j = blockIdx.x*blockDim.x + threadIdx.x;
  int tid = i*N + j;
  if (tid < N*N) {
    S[tid] = A[tid] + B[tid];
  }
}


// Fills a vector with random float entries.
void randomInit(float* data, int N) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      int tid = i*N+j;
      data[tid] = (float)drand48();
    }
  }
}


int main(int argc, char* argv[])
{

  if (argc != 4) {
    fprintf(stderr, "Syntax: %s <matrix size N> <block size> <device id>\n", argv[0]);
    return EXIT_FAILURE;
  }

  int N = atoi(argv[1]);
  int BlockSize = atoi(argv[2]);
  int devId = atoi(argv[3]);

  checkCuda( hipSetDevice(devId) );

  // set seed for drand48()
  srand48(42);

  // allocate host memory for matrices A and B
  printf("Allocate host memory for matrices A and B...\n");
  float* A = (float*) malloc(N * N * sizeof(float));
  float* B = (float*) malloc(N * N * sizeof(float));
  float* S = (float*) malloc(N * N * sizeof(float));

  // initialize host matrices
  printf("Initialize host matrices...\n");
  randomInit(A, N);
  randomInit(B, N);

  // allocate device matrices (linearized)
  printf("Allocate device matrices (linearized)...\n");
  float* dev_A = NULL; 
  float* dev_B = NULL;
  float* dev_S = NULL;
  checkCuda( hipMalloc((void**) &dev_A, N * N * sizeof(float)) );
  checkCuda( hipMalloc((void**) &dev_B, N * N * sizeof(float)) );
  checkCuda( hipMalloc((void**) &dev_S, N * N * sizeof(float)) );

  // copy host memory to device
  checkCuda( hipMemcpy(dev_A, A, N*N*sizeof(float), hipMemcpyHostToDevice) );
  checkCuda( hipMemcpy(dev_B, B, N*N*sizeof(float), hipMemcpyHostToDevice) );

  // execute the kernel
  printf("Execute the kernel...\n");

  int GridSize = (N + BlockSize-1) / BlockSize;
  dim3 gridDim(GridSize, GridSize);
  dim3 blockDim(BlockSize, BlockSize);

  matSum<<< gridDim, blockDim >>>(dev_S, dev_A, dev_B, N);

  // copy result from device to host
  checkCuda( hipMemcpy( S, dev_S, N * N * sizeof(float),hipMemcpyDeviceToHost) );

  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, devId) );
  printf("Device: %s\n", prop.name);

  // clean up memory
  free(A);
  free(B);
  free(S);
  checkCuda( hipFree(dev_A) );
  checkCuda( hipFree(dev_B) );
  checkCuda( hipFree(dev_S) );

  return 0;
}

